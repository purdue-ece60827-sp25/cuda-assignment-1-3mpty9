#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if(idx < size)
		y[idx] = scale * x[idx] + y[idx];

}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	size_t vectorBytes = vectorSize * sizeof(float);
	
	//	Allocate input vectors h_x and h_y in host memory
	float* h_a = (float*)malloc(vectorBytes);
	float* h_b = (float*)malloc(vectorBytes);
	float* h_c = (float*)malloc(vectorBytes);

	

	if (!h_a || !h_b || !h_c ) {
		printf("Unable to malloc memory ... Exiting!");
		return -1;
	}

	//	Initialize input vectors
	vectorInit(h_a, vectorSize);
	vectorInit(h_b, vectorSize);
	memcpy(h_c, h_b, vectorBytes);

	float scale = 2.0f;

	#ifndef DEBUG_PRINT_DISABLE 
		printf("\n Adding vectors : \n");
		printf(" scale = %f\n", scale);
		printf(" a = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", h_a[i]);
		}
		printf(" ... }\n");
		printf(" b = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", h_b[i]);
		}
		printf(" ... }\n");
	#endif

	//	Allocate vectors in devie memory
	float* d_a;
	hipMalloc(&d_a, vectorBytes);
	float* d_b;
	hipMalloc(&d_b, vectorBytes);

	// Copy vectors from host memory to device memory
	hipMemcpy(d_a, h_a, vectorBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, vectorBytes, hipMemcpyHostToDevice);

	// Invoke kernel
	int threadsPerBlock = 256;
    int blocksPerGrid =
            (vectorSize + threadsPerBlock - 1) / threadsPerBlock;
    saxpy_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, scale, vectorSize);


	// Copy result from device memory to host memory
    // h_C contains the result in host memory
	hipMemcpy(h_c, d_b, vectorBytes, hipMemcpyDeviceToHost);

	#ifndef DEBUG_PRINT_DISABLE 
		printf(" c = { ");
		for (int i = 0; i < 5; ++i) {
			printf("%3.4f, ", h_c[i]);
		}
		printf(" ... }\n");
	#endif

	int errorCount = verifyVector(h_a, h_b, h_c, scale, vectorSize);
	std::cout << "Found " << errorCount << " / " << vectorSize << " errors \n";

	// Free device memory
	hipFree(d_a);
    hipFree(d_b);

	// Free host memory
    free(h_a);
    free(h_b);
	free(h_c);

	std::cout << "Lazy, you are!\n";
	std::cout << "Write code, you must\n";

	return 0;
}

/* 
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is responsible for sampleSize points. 
 *pSums is a pointer to an array that holds the number of 'hit' points for each thread. The length of this array is pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx >= pSumSize) return;

	hiprandState_t rng;
	hiprand_init(clock64(), idx, 0, &rng);

	uint64_t hitCount = 0;

	for(uint64_t i = 0; i < sampleSize; i++){
		float x = hiprand_uniform(&rng);
		float y = hiprand_uniform(&rng);
		if(int(x * x + y * y) == 0){
			++ hitCount;
		}
	}
	
	pSums[idx] = hitCount;

}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= reduceSize) return;

    uint64_t sum = 0;
    for (uint64_t i = idx; i < pSumSize; i += reduceSize) {
        sum += pSums[i];
    }
    totals[idx] = sum;
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 0;

	//      Insert code here
	uint64_t *d_pSums;
	size_t pSumsBytes = generateThreadCount * sizeof(uint64_t);
	uint64_t *d_totals;
	size_t totalsBytes = generateThreadCount * sizeof(uint64_t);
	hipMalloc(&d_pSums, pSumsBytes);
	hipMalloc(&d_totals, totalsBytes);

	generatePoints<<<(generateThreadCount + 255) / 256, 256>>>(d_pSums, generateThreadCount, sampleSize);
	reduceCounts<<<(reduceThreadCount + 255) / 256, 256>>>(d_pSums, d_totals, generateThreadCount, reduceSize);

	uint64_t *h_totals = (uint64_t*)malloc(totalsBytes);
	hipMemcpy(h_totals, d_totals, totalsBytes, hipMemcpyDeviceToHost);

	uint64_t totalHitCount = 0;
	for (int i = 0; i < reduceThreadCount; ++i) {
		totalHitCount += h_totals[i];
	}

	// Calculate the approximate value of pi
	approxPi = ((double)totalHitCount / sampleSize) / generateThreadCount;
	approxPi = approxPi * 4.0f;
	// Free allocated memory
	hipFree(d_pSums);
	hipFree(d_totals);
	free(h_totals);

	std::cout << "Sneaky, you are ...\n";
	std::cout << "Compute pi, you must!\n";
	return approxPi;
}
